

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <sys/time.h>

#define BOX_SIZE	23000 /* size of the data box on one dimension            */

/* descriptors for single atom in the tree */
typedef struct atomdesc {
	double x_pos;
	double y_pos;
	double z_pos;
} atom;

typedef struct hist_entry{
	unsigned long long d_cnt;   /* need a long long type as the count might be huge */
} bucket;

bucket *histogram;		/* list of all buckets in the histogram   */
long long PDH_acnt;		/* total number of data points            */
int num_buckets;		/* total number of buckets in the histogram */
double PDH_res;			/* value of w                             */
atom *atom_list;		/* list of all data points                */

struct timezone Idunno;	
struct timeval startTime, endTime;


__device__ double
p2p_distance(atom *atom, int ind1, int ind2) {
	double x1 = atom[ind1].x_pos;
	double x2 = atom[ind2].x_pos;

	double y1 = atom[ind1].y_pos;
	double y2 = atom[ind2].y_pos;

	double z1 = atom[ind1].z_pos;
	double z2 = atom[ind2].z_pos;
		
	return sqrt((x1 - x2)*(x1 - x2) + (y1 - y2)*(y1 - y2) + (z1 - z2)*(z1 - z2));
}


double report_running_time() {
	long sec_diff, usec_diff;
	gettimeofday(&endTime, &Idunno);
	sec_diff = endTime.tv_sec - startTime.tv_sec;
	usec_diff= endTime.tv_usec-startTime.tv_usec;
	if(usec_diff < 0) {
		sec_diff --;
		usec_diff += 1000000;
	}
	printf("Running time for CPU version: %ld.%06ld\n", sec_diff, usec_diff);
	return (double)(sec_diff*1.0 + usec_diff/1000000.0);
}

__global__ void 
PDH_baseline(bucket *histogram, atom *atom, double weight, int size) {
	int i, j;
	int position;
	double distance;
	
	// Add the thread Index with the block index and the dim x
	i = blockIdx.x * blockDim.x + threadIdx.x;
	j = i + 1;
	
	// Get the distance and then atomic add the position of the histogram with 1
	for (int a = j; a < size; a++) {
		distance = p2p_distance(atom, i, a);
		position = (int) (distance / weight);
		atomicAdd( &histogram[position].d_cnt, 1);
	}
}

void output_histogram(){
	int i; 
	long long total_cnt = 0;
	for(i=0; i< num_buckets; i++) {
		if(i%5 == 0) /* we print 5 buckets in a row */
			printf("\n%02d: ", i);
		printf("%15lld ", histogram[i].d_cnt);
		total_cnt += histogram[i].d_cnt;
	  	/* we also want to make sure the total distance count is correct */
		if(i == num_buckets - 1)	
			printf("\n T:%lld \n", total_cnt);
		else printf("| ");
	}
}


int main(int argc, char const *argv[])
{
	PDH_acnt = atoi(argv[1]);	// Number of atoms
	PDH_res = atof(argv[2]);	// Input Distance: W
	num_buckets = (int)(BOX_SIZE * 1.732 / PDH_res) + 1;
	size_t histogramSize = sizeof(bucket)*num_buckets;
	size_t atomSize = sizeof(atom)*PDH_acnt;
	histogram = (bucket *)malloc(histogramSize);
	atom_list = (atom *)malloc(atomSize);
	bucket *d_histogram = NULL;
	atom *d_atom_list = NULL;

	srand(1);
	/* generate data following a uniform distribution */
	for(int i = 0;  i < PDH_acnt; i++) {
		atom_list[i].x_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].y_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
		atom_list[i].z_pos = ((double)(rand()) / RAND_MAX) * BOX_SIZE;
	}

	// Allocate memory to GPU arrays and then copy the data from the CPU arrays
	hipMalloc((void**) &d_histogram, histogramSize);
	hipMalloc((void**) &d_atom_list, atomSize);
	hipMemcpy(d_histogram, histogram, histogramSize, hipMemcpyHostToDevice);
	hipMemcpy(d_atom_list, atom_list, atomSize, hipMemcpyHostToDevice);

	/* start counting time */
	gettimeofday(&startTime, &Idunno);

	// Launch the kernal and perform calcualtions with the GPU PDH_baseline
	PDH_baseline <<<ceil(PDH_acnt/32), 32>>> (d_histogram, d_atom_list, PDH_res, PDH_acnt);
	hipMemcpy(histogram, d_histogram, histogramSize, hipMemcpyDeviceToHost);

	/* check the total running time */ 
	report_running_time();

	// Print the histogram
	output_histogram();

	// Free the GPU(device) and the CPU(host) arrays
	hipFree(d_histogram);
	hipFree(d_atom_list);
	free(histogram);
	free(atom_list);

	return 0;
}
